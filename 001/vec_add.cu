
#include <hip/hip_runtime.h>


__global__ void vecAddKernel(float* A, float* B, float*C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

void vecAdd(float* A_h, float* B_h, float* C_h, int n) {
    int size = n * sizeof(float);
    float *A_d, *B_d, *C_d;

    // Part 1: Allocate device memory for A, B, and C
    hipError_t err = hipMalloc((void**)&A_d, size);
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
        exit(EXIT_FAILURE);
    }

    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);
    // Copy A and B to device memory
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    // Part 2: Call kernel – to launch a grid of threads
    // to perform the actual vector addition
    vecAddKernel<<<ceil(n / 256.0), 256>>>(A_d, B_d, C_d, n);

    // Part 3: Copy C from the device memory
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    // Free device vectors
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}