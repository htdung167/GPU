
#include <hip/hip_runtime.h>


void vecAdd(float* A_h, float* B_h, float* C_h, int n) {
    int size = n * sizeof(float);
    float *A_d, *B_d, *C_d;

    // Part 1: Allocate device memory for A, B, and C
    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);
    // Copy A and B to device memory
    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    // Part 2: Call kernel – to launch a grid of threads
    // to perform the actual vector addition

    // Part 3: Copy C from the device memory
    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    // Free device vectors
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}